
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
// CUDA kernel function
__global__
void add(int n, float *x, float *y) {
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void) {
    int N = 1<<2; // 1M elements

    float *x, *y;
    hipMalloc(&x, N*sizeof(float));
    hipMalloc(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    /*
    // Run kernel on 1M elements on the CPU
    add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host/cpu.
    // Because GPU kernel launches don't block the cpu thread.
    cudaDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    */
    // Free memory
    //delete [] x;
    //delete [] y;
    hipFree(x);
    hipFree(y);

    return 0;
}
